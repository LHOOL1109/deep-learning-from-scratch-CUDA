#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <core/tensor.cuh>

Tensor matmul(const Tensor& A, const Tensor& B, hipblasHandle_t handle)
{
    // A: (M, K), B: (K, N)
    int K = A.width();
    int M = A.size() / K;
    int N = B.width();

    Tensor C(M, N);

    const float alpha = 1.0f;
    const float beta = 0.0f;

    hipblasSgemm(
        handle,
        HIPBLAS_OP_N, HIPBLAS_OP_N,
        N, M, K,
        &alpha,
        B.device_ptr(), N,
        A.device_ptr(), K,
        &beta,
        C.device_ptr(), N
    );
    return C;
}
